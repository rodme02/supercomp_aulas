#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <iostream>
#include <cmath>
#include <chrono>  // Para medir o tempo de execução

// Functor para calcular a diferença ao quadrado em relação à média
struct variance_op {
    float mean;
    variance_op(float mean_) : mean(mean_) {}

    __host__ __device__
    float operator()(const float& x) const {
        float diff = x - mean;
        return diff * diff;
    }
};

// Função para calcular a variância usando a técnica de Fusion Kernel
float calculate_variance_fusion_kernel(const thrust::device_vector<float>& d_vec, float mean) {
    return thrust::transform_reduce(d_vec.begin(), d_vec.end(), variance_op(mean), 0.0f, thrust::plus<float>()) / d_vec.size();
}

// Função para calcular a variância em etapas separadas
float calculate_variance_separate_steps(const thrust::device_vector<float>& d_vec) {
    // Etapa 1: Cálculo da média
    float mean = thrust::reduce(d_vec.begin(), d_vec.end(), 0.0f, thrust::plus<float>()) / d_vec.size();

    // Etapa 2: Cálculo das diferenças ao quadrado e soma
    thrust::device_vector<float> diff_squares(d_vec.size());
    thrust::transform(d_vec.begin(), d_vec.end(), diff_squares.begin(), variance_op(mean));

    // Reduzir para obter a soma das diferenças ao quadrado
    float variance = thrust::reduce(diff_squares.begin(), diff_squares.end(), 0.0f, thrust::plus<float>()) / d_vec.size();
    
    return variance;
}

int main() {
    const int sizes[] = {1000000, 5000000, 10000000};  // Testar com tamanhos maiores

    for (int size : sizes) {
        thrust::host_vector<float> h_v(size);
        for (int i = 0; i < size; i++) {
            h_v[i] = rand() % 100;
        }
        thrust::device_vector<float> d_v = h_v;

        std::cout << "Tamanho do vetor: " << size << std::endl;

        // 1. Cálculo da variância usando Fusion Kernel
        auto start_fusion = std::chrono::high_resolution_clock::now();
        hipDeviceSynchronize();  // Sincronizar antes da medição
        float mean_fusion = thrust::reduce(d_v.begin(), d_v.end(), 0.0f, thrust::plus<float>()) / d_v.size();
        float variance_fusion = calculate_variance_fusion_kernel(d_v, mean_fusion);
        hipDeviceSynchronize();  // Garantir que o cálculo terminou
        auto end_fusion = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> fusion_time = end_fusion - start_fusion;

        std::cout << "  [Fusion Kernel] Variância: " << variance_fusion << ", Tempo: " << fusion_time.count() << " segundos" << std::endl;

        // 2. Cálculo da variância em etapas separadas
        auto start_separate = std::chrono::high_resolution_clock::now();
        hipDeviceSynchronize();  // Sincronizar antes da medição
        float variance_separate = calculate_variance_separate_steps(d_v);
        hipDeviceSynchronize();  // Garantir que o cálculo terminou
        auto end_separate = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> separate_time = end_separate - start_separate;

        std::cout << "  [Etapas Separadas] Variância: " << variance_separate << ", Tempo: " << separate_time.count() << " segundos" << std::endl;

        std::cout << std::endl;
    }

    return 0;
}
