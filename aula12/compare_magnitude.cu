#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <cmath>
#include <iostream>
#include <chrono>
#include <vector>

struct square {
    __host__ __device__
    float operator()(const float& x) const {
        return x * x;
    }
};

float magnitude_gpu(thrust::device_vector<float>& v) {
    float sum_of_squares = thrust::transform_reduce(v.begin(), v.end(), square(), 0.0f, thrust::plus<float>());
    return std::sqrt(sum_of_squares);
}

float magnitude_cpu(const std::vector<float>& v) {
    float sum_of_squares = 0.0f;
    for (const float& x : v) {
        sum_of_squares += x * x;
    }
    return std::sqrt(sum_of_squares);
}

int main() {
    // Gerando vetor aleatório
    const int N = 1000000;

    std::vector<float> h_v(N);
    for (int i = 0; i < N; i++) {
        h_v[i] = rand() % 100;
    }

    thrust::device_vector<float> d_v(h_v);

    // Cálculo na CPU
    auto start_cpu = std::chrono::high_resolution_clock::now();
    float mag_cpu = magnitude_cpu(h_v);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_time = end_cpu - start_cpu;

    // Cálculo na GPU
    auto start_gpu = std::chrono::high_resolution_clock::now();
    float mag_gpu = magnitude_gpu(d_v);
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_time = end_gpu - start_gpu;

    std::cout << "Magnitude (CPU): " << mag_cpu << " em " << cpu_time.count() << " segundos\n";
    std::cout << "Magnitude (GPU): " << mag_gpu << " em " << gpu_time.count() << " segundos\n";

    return 0;
}
